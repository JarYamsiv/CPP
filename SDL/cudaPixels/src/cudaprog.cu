
#include <hip/hip_runtime.h>
__global__ void setpix(int **pix,int *t)
{
	pix[0][0] = t[0];
}

class CUDA
{
public:
	CUDA(int _width,int _height)
	{
		width = _width;
		height = _height;
		pixels = new int[width*height];
		int size = width*height*sizeof(int);
		hipMalloc((void**)&pixels,size);
	}
	~CUDA()
	{
		hipFree(pixels);
		delete [] pixels;
	}
private:
	int width,height;
	int *pixels;
};